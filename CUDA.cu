// Author's : Rakesh Ginjupalli, Felix Rohrer
// Date		: 12/27/09
// We thank Dr Gaurav Khanna for his support


// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>

// Variables
float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

int iNumElements = 10;	// Length of float arrays to process
bool DEBUG = true;

// Functions
void fillFloatArray(float* arr, int length);
void printFloatArray(float* arr, char* name, int length);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main(int argc, char **argv)
{
	// set amount of numbers to be calculated
	if(argc > 1){
		iNumElements = atoi(argv[1]);
		printf("Setting numbers to %d\n", iNumElements);
		if(argc == 3){
			DEBUG = false;
		}
	}

    size_t size = iNumElements * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize input vectors
    fillFloatArray(h_A, iNumElements);
    fillFloatArray(h_B, iNumElements);

	if(DEBUG){
		printFloatArray(h_A, "Array A",  iNumElements);
		printFloatArray(h_B, "Array B",  iNumElements);
	}

    // Allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (iNumElements + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, iNumElements);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
	if(DEBUG){
		printFloatArray(h_C, "Result",  iNumElements);
	}

    // Free device memory
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    // Free host memory
    if (h_A) free(h_A);
    if (h_B) free(h_B);
    if (h_C) free(h_C);
}

void fillFloatArray(float* arr, int length){
	for(int i=0;i<length;i++){
		arr[i] = rand() / (float)RAND_MAX;
	}
}

void printFloatArray(float* arr, char* name, int length){
	printf("%s:\n", name);
	for(int i=0;i<length;i++){
		printf("%.1f ", arr[i]);
	}printf("\n\n");
}
